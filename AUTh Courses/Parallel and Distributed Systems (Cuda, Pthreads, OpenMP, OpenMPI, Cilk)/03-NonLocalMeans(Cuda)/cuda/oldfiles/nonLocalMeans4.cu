
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
// #include "cuda_runtime.h"
#define NDEBUG1
#include <assert.h>

#define tx threadIdx.x
#define ty threadIdx.y
#define bx blockIdx.x
#define by blockIdx.y

#define NN 64
#define multi 2
#define BSZ0 16
#define BSZ (multi * (BSZ0))
#define GSZ (NN / BSZ)

#define MAX_NSZ 7
#define NSZ 5
#define NSZ2 (NSZ / 2)

#define BSZ_HL (BSZ + NSZ - 1)

#define SER(row, col, ncols) ((row) * (ncols) + (col))

__constant__ float gaussDistW[MAX_NSZ * MAX_NSZ];

// me 4*4 blocks kai kai 1024 thr/block kai gia 4 pixel/thread vgainei ligo panw apo 32k shared
// alla den mporw na exw 1024 thread, mallon giati einai mexri 768.
// ara to spaw se 256/block? -> ipologismos metaforwn.

__device__ __forceinline__ void getSharedBlock(float sharedImg[BSZ_HL][BSZ_HL], const float *globalImg, int I0, int N) {
	int ii = SER(ty, tx, BSZ0); // 2d to 1d index of thread i in the block
	do {
		int I  = ii % BSZ_HL; // x index in block including padding
		int J  = ii / BSZ_HL; // y index in block including padding
		int IGlobal = I0 + SER(J, I, N);      // global input index
		assert(I < BSZ_HL);
		if( (I < BSZ_HL) && (J < BSZ_HL) && (ii < N * N) ) {
			sharedImg[I][J] = globalImg[IGlobal]; // download from global
		}
		ii += BSZ0 * BSZ0; // next iteration starts THREADNUM position after
	} while ( ii < BSZ_HL * BSZ_HL ); // only J check needed ? 
}
__device__ __forceinline__ void getWeight(float blockImg[BSZ_HL][BSZ_HL], float foreignBlockImg[BSZ_HL][BSZ_HL], float sigma, float weightSum[multi * multi], float fSum[multi * multi]) {
	// Compute block weights with self
	// new tx is (1) blockdim away
	#define txM (tx + ( mu % multi ) * blockDim.x)
	#define tyM (ty + ( mu / multi ) * blockDim.y)
	for (int mu = 0; mu < multi * multi; mu++) { // Multiple pixels per thread
		for(int k = 0; k < BSZ; k++ ) { // Other block
			for(int l = 0; l < BSZ; l++ ) {
				float partialW = 0;
				for(int m = -NSZ2; m <= NSZ2; m++) // Neighbourhoud
					for(int n = -NSZ2; n <= NSZ2; n++)
						partialW += gaussDistW[SER((n + MAX_NSZ / 2), (m + MAX_NSZ / 2), MAX_NSZ)]
							* powf( ( blockImg[(txM + NSZ2) + m][(tyM + NSZ2) + n] 
							 - foreignBlockImg[(k   + NSZ2) + m][(l   + NSZ2) + n] ), 2);
				// if (!txM && !tyM && k==1) printf("%f\n",partialW);
				partialW = expf((-partialW / sigma));
				weightSum[mu] += partialW;
				fSum[mu]      += partialW * foreignBlockImg[k + NSZ2][l + NSZ2];
			}
		}
	}
}

__device__ __forceinline__ void downloadAndCalculate(float blockImg[BSZ_HL][BSZ_HL], float foreignBlockImg[BSZ_HL][BSZ_HL],
	const float *inputImg, float sigma, float *weightSum, float *fSum, int N, int I0) {
		getSharedBlock(foreignBlockImg, inputImg, I0, N);
		__syncthreads();
		getWeight(blockImg, foreignBlockImg, sigma, weightSum, fSum);
}


__global__ void nlm(float const *inputImg, float *outputImg, int N, float sigma) {
	assert(NN == N);
	int N2 = N + NSZ - 1; // input image with padding
	assert(GSZ == gridDim.x);
	assert(BSZ0 == blockDim.x);
	__shared__ float        blockImg[BSZ_HL][BSZ_HL];
	__shared__ float foreignBlockImg[BSZ_HL][BSZ_HL];
	
	// if(!tx && !ty && !bx && !by) {
	// for (int mu = 0; mu < 49; mu++) {
	// 	printf("%f ",gaussDistW[mu]);
	// }
	// }
	// __syncthreads();

	float weightSum[multi * multi], fSum[multi * multi]; // Weightsums for multiple pixels per thread.
	for (int mu = 0; mu < multi * multi; mu++) { // Multiple pixels per thread
		weightSum[mu] = 0;
		fSum[mu] = 0;
	}

	// put inside
	int I0 = SER(by * BSZ, bx * BSZ, N2); // Download this block's pixels
	downloadAndCalculate(blockImg, blockImg, inputImg, sigma, weightSum, fSum, N2, I0);
	for (char i = 0; i < GSZ; i++) { // gia kathe BLOCK stin arxiki eikona (X)
		for (char j = 0; j < GSZ; j++) {
			if ( !(by == j && bx == i) ) {
				// Download other blocks
				int I1 = SER(j * BSZ, i * BSZ, N2); //first pixel in block. Used as a ref point to calculate the block.(pg21)
				downloadAndCalculate(blockImg, foreignBlockImg, inputImg, sigma, weightSum, fSum, N2, I1);
			}
		}
	}

	for (int mu = 0; mu < multi * multi; mu++) { // Multiple pixels per thread
		// add NSZ2 to skip the padding pixels
		outputImg[SER(by * BSZ, bx * BSZ, N) + SER(tyM, txM, N)] = fSum[mu] / weightSum[mu];
		//blockImg[(txM + NSZ2)][(tyM + NSZ2)]
		//inputImg[I0 + SER(NSZ2, NSZ2, N2) + SER(tyM, txM, N)]
	}
}

// template __global__ void kernel<false>();